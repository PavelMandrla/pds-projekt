#include "hip/hip_runtime.h"
#include "add.h"
#include "hip/hip_runtime.h"
#include ""

#define RECT_SIZE 32
#define HIST_SIZE 9*256

__device__ bool isPixelLighterThanCentre(int y, int x, unsigned char centerVal, unsigned char* inImg, int width, int height) {
    if (x < 0 || y < 0 || x > width - 1 || y > height - 1) {
        return false;
    }
    int idx = x + y * gridDim.x;
    return inImg[idx] >= centerVal;
}

__device__ unsigned char getLBPVal(int x, int y, unsigned char* inImg, int width, int height) {
    unsigned char result = 0;
    int idx = x + y * gridDim.x;
    unsigned char center = inImg[idx];

    result |= ((unsigned char) isPixelLighterThanCentre(y-1, x-1, center, inImg, width, height)) << 7;
    result |= ((unsigned char) isPixelLighterThanCentre(y-1, x, center,    inImg, width, height)) << 6;
    result |= ((unsigned char) isPixelLighterThanCentre(y-1, x+1, center, inImg, width, height)) << 5;
    result |= ((unsigned char) isPixelLighterThanCentre(y, x+1, center, inImg, width, height)) << 4;
    result |= ((unsigned char) isPixelLighterThanCentre(y+1, x+1, center, inImg, width, height)) << 3;
    result |= ((unsigned char) isPixelLighterThanCentre(y+1, x,   center, inImg, width, height)) << 2;
    result |= ((unsigned char) isPixelLighterThanCentre(y+1, x-1, center, inImg, width, height)) << 1;
    result |= ((unsigned char) isPixelLighterThanCentre(y, x-1, center, inImg, width, height)) << 0;

    return result;

}

__global__ void lbpCUDA(unsigned char* inImg, unsigned char* outImg, int width, int height) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = x + y * gridDim.x;

    outImg[idx] = getLBPVal(x, y, inImg, width, height);
}



__device__ bool getRectCoords(int &xFrom, int&yFrom, int startX, int startY, int width, int height) {
    int areaX, areaY;
    areaX = startX + blockIdx.x;
    areaY = startY;

    while (areaX > width - 256 * 3) {
        areaY++;
        areaX -= 256 * 3;

        if (areaY > height - 256 * 3) {
            xFrom = -1;
            yFrom = -1;
            return false;
        }
    }

    switch (threadIdx.x) {
        case 0:
            xFrom = areaX;
            yFrom = areaY;
            break;
        case 1:
            xFrom = areaX + 256;
            yFrom = areaY;
            break;
        case 2:
            xFrom = areaX + 2 * 256;
            yFrom = areaY;
            break;
        case 3:
            xFrom = areaX;
            yFrom = areaY + 256;
            break;
        case 4:
            xFrom = areaX + 256;
            yFrom = areaY + 256;
            break;
        case 5:
            xFrom = areaX + 2 * 256;
            yFrom = areaY + 256;
            break;
        case 6:
            xFrom = areaX;
            yFrom = areaY + 2 * 256;
            break;
        case 7:
            xFrom = areaX + 256;
            yFrom = areaY + 2 * 256;
            break;
        case 8:
            xFrom = areaX + 2 * 256;
            yFrom = areaY + 2 * 256;
            break;
    }

    return true;
}

__global__ void calculateHistograms(unsigned char* lbpImg, int* histogram, int startX, int startY, int width, int height) {
    int histStart = 9 * 256 * blockIdx.x + 256 * threadIdx.x;
    int xFrom, yFrom;
    getRectCoords(xFrom, yFrom, startX, startY, width, height);

    for (int i = 0; i < 256; i++) {
        histogram[histStart + i] = 0;
    }

    for (int x = 0; x < RECT_SIZE; x++) {
        for (int y = 0; y < RECT_SIZE; y++) {
            int value = lbpImg[x+y*width];
            histogram[value + histStart]++;
        }
    }
}




__global__ void calcuateDistances(int* histograms, int*dataset, double* distances, int datasetSize, int histogramSize, int histOrder) {
    long threadPos = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadPos < histogramSize * datasetSize) {
        int histogramPos = threadPos / datasetSize;
        int datasetPos = threadPos % datasetSize;
        //double distance = 0;
        distances[threadPos] = 0;

        for (int i = 0; i < 9 * 256; i++) {
            distances[threadPos] += pow(
                    (double) (histograms[i + histogramPos * HIST_SIZE]) - dataset[i + datasetPos * HIST_SIZE], 2);
        }
        distances[threadPos] = sqrt(distances[threadPos]);
    }
}

__global__ void getKNNDistance(int neighbourCount, double*histogramDistances, double*knnDistances, int datasetSize, int histogramSize) {
    long threadPos = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadPos < datasetSize * histogramSize) {
        double* nearestNeighbours = new double[neighbourCount];
        for (int i = 0; i < neighbourCount; i++){
            nearestNeighbours[i] = -1;
        }

        for (int i = 0; i < datasetSize; i++) {

            for (int j = 0; j < neighbourCount - 1; j++){
                if (j == 0) {
                    double dst = histogramDistances[i + threadPos * datasetSize]
                    if (nearestNeighbours[j] == -1 || nearestNeighbours[j] > dst) {
                        nearestNeighbours[j] = dst;
                    } else {
                        break;
                    }
                } else {
                    if (nearestNeighbours[j + 1] == -1 || nearestNeighbours[j + 1] > nearestNeighbours[j]) {
                        //switch values
                        double tmp = nearestNeighbours[j + 1];
                        nearestNeighbours[j + 1] = nearestNeighbours[j];
                        nearestNeighbours[j] = tmp;
                    } else {
                        break;
                    }
                }
            }
        }
        knnDistances[threadPos] = 0;
        for (int i = 0; i < neighbourCount; i++){
            knnDistances[threadPos] += nearestNeighbours[i];
        }
    }
}


void convertImageToLBP(unsigned char* imputImg, int width, int height, int* dataset) {
    unsigned char* Dev_InImg = nullptr;
    unsigned char* Dev_OutImg = nullptr;

    hipMalloc((void**)&Dev_InImg,  height*width);
    hipMalloc((void**)&Dev_OutImg, height*width);

    hipMemcpy(Dev_InImg, imputImg, width * height, hipMemcpyHostToDevice);

    dim3 gridImg(width, height);
    lbpCUDA<<<gridImg, 1>>>(Dev_InImg, Dev_OutImg, width, height);

    hipMemcpy(imputImg, Dev_OutImg, width * height, hipMemcpyDeviceToHost);

    hipFree(Dev_InImg);

    int histogramSize = 9 * 256;        //VELIKOST HISTOGRAMU
    int histogramCount = (width - (histogramSize - 1)) * (height - (histogramSize - 1));  //POCET HISTOGRAMU V OBRAZKU
    int* histograms = new int[histogramCount * histogramSize];   //PAMET PRO HISTOGRAMY V POCITACI
    int histGrid = 1000000;            // POCET HISTOGRAMU, KTERE SE BUDOU ZAROVEN POCITAT NA GPU

    int* Dev_histograms = nullptr;
    hipMalloc((void**)&Dev_histograms, histogramSize * histGrid * sizeof(int));

    dim3 gridHist(histGrid, 1);
    dim3 blockHist(9, 1,1); //HISTOGRAM JE SLOZENY Z 9 SUBHISTOGRAMU

    //VYPOCITEJ HISTOGRAMY A ULOZ JE V PAMETI
    int i = 0;
    int* writeFront = histograms;
    while (histGrid * (i + 1) <= histogramCount) {
        //startX = (histGrid * i) % width;
        //startY = (histGrid * i) / width;
        calculateHistograms<<<gridHist, blockHist>>>(Dev_OutImg, Dev_histograms, (histGrid * i) % width, (histGrid * i) / width, width, height);

        hipMemcpy(writeFront, Dev_histograms, histogramSize * histogramCount * sizeof(int), hipMemcpyDeviceToHost);
        writeFront += histGrid * histogramSize;
        i++;
    }
    int restHistGrid = histogramCount - histGrid * i;   //ZBYTEK NEDOPOCITANYCH
    if (restHistGrid > 0) {
        dim3 restGridHist(restHistGrid, 1);
        calculateHistograms<<<gridHist, blockHist>>>(Dev_OutImg, Dev_histograms, (histGrid * i) % width, (histGrid * i) / width, width, height);
        hipMemcpy(writeFront, Dev_histograms, histogramSize * histGrid * sizeof(int), hipMemcpyDeviceToHost);
    }

    hipFree(Dev_histograms);
    hipFree(Dev_OutImg);

    //NAHRAJ DATASET DO GPU
    int* Dev_dataset = nullptr;
    hipMalloc((void**)&Dev_dataset, 5000 * HIST_SIZE * sizeof(int));   //ALOKACE PAMETI PRO DATASET
    hipMemcpy(Dev_dataset, dataset, 5000 * HIST_SIZE * sizeof(int), hipMemcpyHostToDevice);

    int biteSize = 1000;
    histGrid = (int)ceil((biteSize * HIST_SIZE) / 1024);

    double* Dev_distances = nullptr;
    hipMalloc((void**)&Dev_distances, histGrid * 5000 * sizeof(double));   // ALOKACE PAMETI PRO VZDALENOSTI

    double* Dev_knnDistances = nullptr;
    hipMalloc((void**)&Dev_knnDistances, histGrid * sizeof(double));   // ALOKACE PAMETI PRO K-NN VZDALENOSTI




    hipFree(Dev_distances);
    hipFree(Dev_knnDistances);
    hipFree(Dev_dataset);
    delete [] histograms;
}
